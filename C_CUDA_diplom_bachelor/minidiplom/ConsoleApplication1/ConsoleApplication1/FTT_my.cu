#include "Full_header.cuh"
#include <chrono>
using namespace std::chrono;
#define DIM 1024
static unsigned char* dev_Main_Window_picture;
static hipfftDoubleComplex* fftarr;
static double Min = 0, Max = 664000.000000;
int T1(int l)                                     // ����������� ������� l = 2**t
{
    int m = 1;
    int nn = 2;
    for (int i = 1; ; i++) { nn = nn * 2; if (nn > l) { m = i; break; } }
    return m;
}
void MY_BPF( int texWidth_Gal) {
    system_clock::time_point start = system_clock::now();
    double invers = 1.0;
    if (invers == 1.0)
    {
        invers = -1.0;
    }
    // else invers=1.0;
    hipfftDoubleComplex* d_data_buf;
    hipMalloc((void**)&d_data_buf, texWidth_Gal * texWidth_Gal * sizeof(hipfftDoubleComplex));
    hipDeviceSynchronize();

    int N = texWidth_Gal;
    int t = T1(N);

    int* dev_Invers;
    hipMalloc((void**)&dev_Invers, N * sizeof(int));
    hipDeviceSynchronize();
    int size = 1024;
    if (size > texWidth_Gal)
    {
        size = texWidth_Gal;

    }
    dim3    grids3((N) / size, 1);
    dim3    threads3(size, 1);
    invers_array_kernel << <grids3, threads3 >> > (dev_Invers, N, t);

    hipDeviceSynchronize();

    dim3    grids1((N) / 16, (N) / 16);
    dim3    threads1(16, 16);
    invers_zarray_str << < grids1, threads1 >> > (fftarr, d_data_buf, dev_Invers, N);
    hipDeviceSynchronize();
    dim3    grids2(N / N, (N) / 16);
    dim3    threads2(1, 16);
    //dim3 grids2= ((N) / 16, (N) / 16);
    int blockSize = 16;
    int numBlocks = (N + blockSize - 1) / blockSize;
    int numstreams = 1;
    hipStream_t streams[32];
    for (int i = 0; i < numstreams; ++i) {
        hipStreamCreate(&streams[i]);
    }
    for (int i = 0; i < numstreams; i++)
    {
        BPFur_str << <(numBlocks / numstreams), blockSize, 0, streams[i] >> > (
            (d_data_buf + (i * N * blockSize * numBlocks / numstreams)), N, t, invers);
        hipDeviceSynchronize();
    }
    // ������������� ������� ������
    for (int i = 0; i < numstreams; ++i) {
        hipStreamSynchronize(streams[i]);
    }
    invers_zarray_collum << < grids1, threads1 >> > (d_data_buf, fftarr, dev_Invers, N);
    for (int i = 0; i < 1; i++)
    {
        BPFur_collum << <numBlocks /*/ numstreams*/, blockSize/*, 0, streams[i]*/ >> > (fftarr/* + (i * N * blockSize * numBlocks / numstreams))*/, N, t, invers);
        hipDeviceSynchronize();
    }
    //  ������������� ������� ������
    for (int i = 0; i < numstreams; ++i) {
        hipStreamSynchronize(streams[i]);
    }
    // ����������� ������� ������
    for (int i = 0; i < numstreams; ++i) {
        hipStreamDestroy(streams[i]);
    }
    system_clock::time_point end = system_clock::now();
    hipFree(d_data_buf);
    hipFree(dev_Invers);
    std::chrono::duration<double, std::milli> duration = end - start;
    printf("cufft %5f millisec\n", duration.count());
}

extern "C" void FFT_MY(hipfftDoubleComplex * FFT, unsigned char* dev_a, int width_A)
{
    fftarr = FFT;
    //unsigned char* dev_Main_Window_picture;
    //unsigned char* fftarray;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float milliseconds_1 = 0, milliseconds_2 = 0, milliseconds_3 = 0, milliseconds_4 = 0;

    hipEventRecord(start);
    hipError_t cudaStatus;
    static int k = 0;
    static char filepath[256];

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds_1, start, stop);
    printf("\n/////////\nload img to aplication : %.5f millisec\n", milliseconds_1);


    hipEventRecord(start);
    cudaStatus = hipMalloc((void**)&dev_Main_Window_picture, width_A * width_A * sizeof(unsigned char) * 1);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error1;

    }
    cudaStatus = hipMemcpy(dev_Main_Window_picture, dev_a, width_A * width_A * sizeof(unsigned char) * 1, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        // goto Error1;
    }
    
    int thread = 0;
    if (width_A < 1024)
    {
        thread = width_A;
    }
    else
    {
        thread = 1024;
    }

    dim3    grids((width_A + 32 - 1) / 32, (width_A + 32 - 1) / 32);
    dim3    threads(32, 32);

    /*Img_to_Complex << <grids, threads >> > (zarray, dev_Main_Window_picture, texWidth);*/
    Img_to_cufftComplex << <grids, threads >> > (fftarr, dev_Main_Window_picture, width_A);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds_2, start, stop);
    printf("create\load to complex : %.5f millisec\n", milliseconds_2);

    //hipEventRecord(start);
    MY_BPF(width_A);

    //hipEventRecord(stop);
    //hipEventSynchronize(stop);
    //hipEventElapsedTime(&milliseconds_3, start, stop);
    //printf("BPF : %.5f millisec\n", milliseconds_3);

    //hipEventRecord(start);
    ///*   stbi_image_free(pixels_Gal);*/
    //dim3 grids1(DIM / 16, DIM / 16);
    //dim3 threads1(16, 16);
    ////kernel << <grids1, threads1 >> > (pixels, ticks, texWidth_A, texHeight_A, texWidth_B, texHeight_B, dev_a, dev_b);


    //DrawPicABSAfterBPFRange << <grids1, threads1 >> > (bufferIMG, fftarray, width_A, Min, Max);
    //hipEventRecord(stop);
    //hipEventSynchronize(stop);
    //hipEventElapsedTime(&milliseconds_4, start, stop);
    //printf("Pan/crop/scaling : %.5f millisec\n", milliseconds_4);
    //printf("full time without draw : %.5f millisec\n", milliseconds_4 + milliseconds_3 + milliseconds_2 + milliseconds_1);
    //Error1:
   // stbi_image_free(dev_a);
    //hipFree(fftarr);
    hipFree(dev_Main_Window_picture);

}