#include "hip/hip_runtime.h"
#include "Full_header.cuh"
__global__ void invers_zarray_collum(hipfftDoubleComplex* zarray, hipfftDoubleComplex* zarray_buf, int* dev_invers, int width)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    //uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
    //uint32_t rev = reverse_bits_gpu(i * 2);
    //rev = rev >> (32 - 12);
    //uint32_t rev1 = reverse_bits_gpu(i * 2 + 1);
    //rev1 = rev1 >> (32 - 12);
    //hipfftDoubleComplex ip, ip2;
    if ((x < width) && (y < width))
    {

        //ip = zarray[x + y * width];
        //ip2= zarray[dev_invers[x] + y * width];
        //zarray[dev_invers[x] + y * width] = ip;
        zarray_buf[x + y * width] = zarray[x + dev_invers[y] * width];
        if (x < width / 2) {
            //                       if (dev_invers[x * 2] != rev || dev_invers[x * 2 + 1] != rev1)
            //                       {
            ///*                           printf("x= %d dev_invers[x]= %d  %d\n", x * 2, dev_invers[x * 2], rev);
            //                           printf("x= %d dev_invers[x]= %d  %d\n", x * 2 + 1, dev_invers[x * 2 + 1], rev1);*/
            //                       }
        }
    }
}