#include "hip/hip_runtime.h"
#include "Full_header.cuh"
#include <chrono>
using namespace std::chrono;
static double Min = 0, Max = 664000.000000;
extern "C" void CUFFTTEST (uchar4 * bufferIMG, unsigned char* dev_a, int width_A) {
    int DIM = 1024;
    static hipfftDoubleComplex *fftarray;
    static unsigned char *dev_Main_Window_picture;
    hipMalloc((void**)&fftarray, width_A * width_A * sizeof(hipfftDoubleComplex));
    hipMalloc((void**)&dev_Main_Window_picture, width_A * width_A * sizeof(unsigned char) * 1);
    hipMemcpy(dev_Main_Window_picture, dev_a, width_A * width_A * sizeof(unsigned char) * 1, hipMemcpyHostToDevice);
    dim3    grids((width_A + 32 - 1) / 32, (width_A + 32 - 1) / 32);
    dim3    threads(32, 32);
    Img_to_cufftComplex << <grids, threads >> > (fftarray, dev_Main_Window_picture, width_A);
    hipfftHandle plan;
    int n[2] = { width_A, width_A };
    hipfftPlanMany(&plan, 2, n, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, 1);
  /*  hipEvent_t start, stop;*/
    float gpuTime = 0.0f;
 /*   hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);*/
    system_clock::time_point start = system_clock::now();
    hipfftExecZ2Z(plan, fftarray, fftarray, HIPFFT_FORWARD);
    hipDeviceSynchronize();
    system_clock::time_point end = system_clock::now();
    //hipDeviceSynchronize();
   
    // ��� �����������
    std::chrono::duration<double, std::milli> duration = end - start;
    printf("cufft %5f millisec\n", duration.count());
    //hipDeviceSynchronize();
    //hipEventRecord(stop, 0);
    //hipEventSynchronize(stop);//����� ���������

    //hipEventElapsedTime(&gpuTime, start, stop);//���������� ���������

    //printf("CUfFt : %.5f millisec\n", gpuTime);

    dim3 grids1(DIM / 16, DIM / 16);
    dim3 threads1(16, 16);
    //kernel << <grids1, threads1 >> > (pixels, ticks, texWidth_A, texHeight_A, texWidth_B, texHeight_B, dev_a, dev_b);


    DrawPicABSAfterBPFRange << <grids1, threads1 >> > (bufferIMG, fftarray, width_A, Min, Max);
    hipfftDestroy(plan);
    //hipEventDestroy(start);
    //hipEventDestroy(stop);
    hipFree(fftarray);
}