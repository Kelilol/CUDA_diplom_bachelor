#include "hip/hip_runtime.h"
#include "Full_header.cuh"
__global__ void invers_array_kernel(int* dev_invers, int n, int t)
{

    int ii = blockIdx.x * blockDim.x + threadIdx.x;
    if (ii < n) {
        int k = 1;
        int k1 = k << (t - 1);
        int b1 = 0;
        for (int i = 1; i <= t / 2 + 1; i++)
        {
            if ((ii & k) != 0) b1 = b1 | k1;
            if ((ii & k1) != 0) b1 = b1 | k;
            k = k << 1;
            k1 = k1 >> 1;
        }
        dev_invers[ii] = b1;
    }
}