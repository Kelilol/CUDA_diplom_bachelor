#include "hip/hip_runtime.h"
#include "Full_header.cuh"
__global__ void BPFur_collum(hipfftDoubleComplex* zarrayPhur, int width, int powerOfTwo, double invers)
{
    int i;
    int  j, ip, l;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int power_SH = powerOfTwo;
    int width_SH = width;


    //int offset = ((idx)*width_SH);

    if (idx < width_SH) {
        hipfftDoubleComplex w, u, t;


        for (l = 1; l <= power_SH; l++)//l �������� ������ k ���������� ����� ��������
        {

            int ll = 1 << l/*2^k */, ll1 = ll >> 1;//(k-1) ����������� ������ ����� pi � �������
            u.x = (1.0);
            u.y = (0.0);
            int mh = 1 << (l - 1);

            w = make_hipDoubleComplex(cos(3.1415926535897932384626433832795 / mh), invers * sin(3.1415926535897932384626433832795 / mh));//������ ����� ���������� �� �������� �������� l=k



            for (j = 1; j <= ll1; j++)//j <= 2^(k-1)
            {
                for (i = j - 1; i < width_SH; i = i + ll)
                {

                    ip = (ll1 + i) * 4096 + idx;

                    t = hipCmul(zarrayPhur[ip], u);
                    zarrayPhur[ip] = hipCsub(zarrayPhur[(i * 4096) + idx], t);
                    zarrayPhur[(i) * 4096 + idx] = hipCadd(zarrayPhur[(i) * 4096 + idx], t);

                }

                u = hipCmul(u, w); //������ ��������� � ��������� p

            }

        }
    }

}