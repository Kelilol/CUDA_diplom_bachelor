#include "hip/hip_runtime.h"
#include "Full_header.cuh"
__global__ void Img_to_cufftComplex(hipfftDoubleComplex* zarray, unsigned char* dev_Main_Window_picture, int width_A)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    if (offset < width_A * width_A)
    {
        zarray[offset].x = (double)dev_Main_Window_picture[(x + y * width_A) * 1];
        zarray[offset].y = 0.0;
    }
}