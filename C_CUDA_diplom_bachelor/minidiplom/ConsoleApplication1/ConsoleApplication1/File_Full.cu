#include "hip/hip_runtime.h"
#include "Full_header.cuh"
#include <stdio.h>
#include <math.h>
#include <chrono>
using namespace std::chrono;
#define DIM 1024
static unsigned char* dev_Main_Window_picture;
hipfftDoubleComplex* fftarray;
static double Min = 0, Max = 664000.000000;

__global__ void drawloadbuff(uchar4* bufferIMG,int width_A)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    if (offset < DIM * DIM)
    {


        bufferIMG[DIM * DIM - 1 - offset].x = 70;
        bufferIMG[DIM * DIM - 1 - offset].y = 255;
        bufferIMG[DIM * DIM - 1 - offset].z = 255;
        bufferIMG[DIM * DIM - 1 - offset].w = 255;
    }
}
__global__ void cudaAddd(int a, int b, int* c) {
    *c = a + b;
}
extern "C" void cudaAdd(uchar4 * bufferIMG, int width_A) {
    int* k;
    hipMalloc((void**)&k, 100);
    
    dim3    grids1((width_A) / 16, (width_A) / 16);
    dim3    threads1(16, 16);
    drawloadbuff << <grids1, threads1 >> > (bufferIMG, width_A);

    hipFree (k);

  
}
__global__ void drawloadbufff(uchar4* bufferIMG, int width_A)
{
    
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    if (offset < DIM * DIM)
    {


        bufferIMG[DIM * DIM - 1 - offset].x = 0;
        bufferIMG[DIM * DIM - 1 - offset].y = 0;
        bufferIMG[DIM * DIM - 1 - offset].z = 255;
        bufferIMG[DIM * DIM - 1 - offset].w = 255;
    }
}

extern "C" void cudaAdd1(uchar4 * bufferIMG, int width_A) {
    int* s;
    hipMalloc((void**)&s, 1000);


    dim3    grids1((width_A) / 16, (width_A) / 16);
    dim3    threads1(16, 16);
    drawloadbufff << <grids1, threads1 >> > (bufferIMG, width_A);



    hipFree(s);


}
__global__ void Draw(uchar4* bufferIMG, unsigned char* zarray, int width_A)
{
    int l = width_A / (DIM);
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x1 = (DIM - x-1) * l;
    int y1 = y * l;
    int offset1 = x1 + y1 * width_A;
    int offset = x + y * blockDim.x * gridDim.x;
    int MAG = 0;
    if (offset < DIM * DIM)
    {
        for (int i = 0; i < l; i++)
        {
            for (int j = 0; j < l; j++)
            {
                MAG += zarray[x1 + j + (y1 + i) * width_A];
            }
        }
        MAG = MAG / (l * l);
        unsigned char mag;
        if (MAG > 255)
        {
            mag = 255;
        }else mag = (unsigned char)MAG;
        bufferIMG[1048575- offset].x = mag;
        bufferIMG[1048575 - offset].y = mag;
        bufferIMG[1048575 - offset].z = mag;
        bufferIMG[1048575 - offset].w = 255;
    }
}

extern "C" void drawloadimg (uchar4* bufferIMG, unsigned char* zarray, int width_A) {
    hipError_t cudaStatus;
    unsigned char* dev_Main_Window_picture1;
    printf("s ");
    hipMalloc((void**)&dev_Main_Window_picture1, width_A * width_A * sizeof(unsigned char) * 1);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error1;

    }
    hipMemcpy(dev_Main_Window_picture1, zarray, width_A * width_A * sizeof(unsigned char) * 1, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        // goto Error1;
    }
    int powerthreadMax = log2(1024);
    int girdthreadmax = pow(2.0, powerthreadMax / 2);
    int powertexWidth = log2(4096);
    int girdtexWidth = pow(2.0, powertexWidth / 2);
    int thread;
    if (girdthreadmax < girdtexWidth)
    {
        thread = girdthreadmax;
    }
    else
    {
        thread = girdtexWidth;
    }
    dim3    grids((1024 + thread - 1) / thread, (1024 + thread - 1) / thread);
    dim3    threads(thread, thread);


 
    Draw << <grids, threads >> > (bufferIMG, dev_Main_Window_picture1, width_A);
    hipFree(dev_Main_Window_picture1);
}





//__device__ uint32_t reverse_bits_gpu(uint32_t x)
//{
//    x = ((x & 0xaaaaaaaa) >> 1) | ((x & 0x55555555) << 1);
//    x = ((x & 0xcccccccc) >> 2) | ((x & 0x33333333) << 2);
//    x = ((x & 0xf0f0f0f0) >> 4) | ((x & 0x0f0f0f0f) << 4);
//    x = ((x & 0xff00ff00) >> 8) | ((x & 0x00ff00ff) << 8);
//    return (x >> 16) | (x << 16);
//}






__global__ void DrawPicABSAfterBPFRange(uchar4* bufferIMG, hipfftDoubleComplex* zarray, int width_A, double min, double max)
{
    int l = width_A / (DIM);
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x1 = (DIM - x-1) * l;
    int y1 = y * l;
    int offset1 = x1 + y1 * width_A;
    int offset = x + y * blockDim.x * gridDim.x;
    double MAG = 0;
    if (offset < DIM * DIM)
    {
        for (int i = 0; i < l; i++)
        {
            for (int j = 0; j < l; j++)
            {
                MAG += hipCabs(zarray[x1 + j + (y1 + i) * width_A]);
            }
        }
        MAG = MAG / (l * l);
        unsigned char mag = 0;
        if (MAG > max) { mag = 255; }
        else {
            mag = (MAG - min) / (max - min) * 255;
        }
        bufferIMG[DIM * DIM - 1 - offset].x = mag;
        bufferIMG[DIM * DIM - 1 - offset].y = mag;
        bufferIMG[DIM * DIM - 1 - offset].z = mag;
        bufferIMG[DIM * DIM - 1 - offset].w = mag;

    }
}
__global__ void Draw(unsigned char* bufferIMG, unsigned char* zarray, int width_A)
{
    int l = width_A / (DIM);
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x1 = (DIM - x) * l;
    int y1 = y * l;
    int offset1 = x1 + y1 * width_A;
    int offset = x + y * blockDim.x * gridDim.x;
    int MAG = 0;
    if (offset < DIM * DIM)
    {
        for (int i = 0; i < l; i++)
        {
            for (int j = 0; j < l; j++)
            {
                MAG += zarray[x1 + j + (y1 + i) * width_A];
            }
        }
        MAG = MAG / (l * l);
        unsigned char mag = (unsigned char)MAG;
        bufferIMG[DIM * DIM - 1 - offset] = mag;
    }
}
__global__ void drawloadbuff(uchar4* bufferIMG, unsigned char* zarray, int width_A)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    if (offset < DIM * DIM)
    {


        bufferIMG[DIM * DIM - 1 - offset].x = zarray[offset];
        bufferIMG[DIM * DIM - 1 - offset].y = zarray[offset];
        bufferIMG[DIM * DIM - 1 - offset].z = zarray[offset];
        bufferIMG[DIM * DIM - 1 - offset].w = 255;
    }
}
int T(int l)                                     // ����������� ������� l = 2**t
{
    int m = 1;
    int nn = 2;
    for (int i = 1; ; i++) { nn = nn * 2; if (nn > l) { m = i; break; } }
    return m;
}
void MYBPF(uchar4* bufferIMG,  int texWidth_Gal) {
    system_clock::time_point start = system_clock::now();
    double invers= 1.0;
    if (invers == 1.0)
    {
        invers = -1.0;
    }
    // else invers=1.0;
    hipfftDoubleComplex* d_data_buf;
    hipMalloc((void**)&d_data_buf, texWidth_Gal * texWidth_Gal * sizeof(hipfftDoubleComplex));
    hipDeviceSynchronize();

    int N = texWidth_Gal;
    int t = T(N);

    int* dev_Invers;
    hipMalloc((void**)&dev_Invers, N * sizeof(int));
    hipDeviceSynchronize();
    int size = 1024;
    if (size > texWidth_Gal)
    {
        size = texWidth_Gal;

    }
    dim3    grids3((N) / size, 1);
    dim3    threads3(size, 1);
    invers_array_kernel << <grids3, threads3 >> > (dev_Invers, N, t);

    hipDeviceSynchronize();

    dim3    grids1((N) / 16, (N) / 16);
    dim3    threads1(16, 16);
    invers_zarray_str << < grids1, threads1 >> > (fftarray, d_data_buf, dev_Invers, N);
    hipDeviceSynchronize();
    dim3    grids2(N / N, (N) / 16);
    dim3    threads2(1, 16);
    //dim3 grids2= ((N) / 16, (N) / 16);
    int blockSize = 16;
    int numBlocks = (N + blockSize - 1) / blockSize;
    int numstreams = 1;
    hipStream_t streams[32];
    for (int i = 0; i < numstreams; ++i) {
        hipStreamCreate(&streams[i]);
    }
    for (int i = 0; i < numstreams; i++)
    {
        BPFur_str << <(numBlocks / numstreams), blockSize, 0, streams[i] >> > ((d_data_buf + (i * N * blockSize * numBlocks / numstreams)), N, t, invers);
        hipDeviceSynchronize();
    }
    // ������������� ������� ������
    for (int i = 0; i < numstreams; ++i) {
        hipStreamSynchronize(streams[i]);
    }
    invers_zarray_collum << < grids1, threads1 >> > (d_data_buf, fftarray, dev_Invers, N);
    for (int i = 0; i < 1; i++)
    {
        BPFur_collum << <numBlocks /*/ numstreams*/, blockSize/*, 0, streams[i]*/ >> > (fftarray/* + (i * N * blockSize * numBlocks / numstreams))*/, N, t, invers);
        hipDeviceSynchronize();
    }
    //  ������������� ������� ������
    for (int i = 0; i < numstreams; ++i) {
        hipStreamSynchronize(streams[i]);
    }
    // ����������� ������� ������
    for (int i = 0; i < numstreams; ++i) {
        hipStreamDestroy(streams[i]);
    }
    system_clock::time_point end = system_clock::now();
    hipFree(d_data_buf);
    hipFree(dev_Invers);
    std::chrono::duration<double, std::milli> duration = end - start;
    printf("cufft %5f millisec\n", duration.count());
}

extern "C" void Test_My_BPF(uchar4* bufferIMG, unsigned char* dev_a, int width_A)
{
    //unsigned char* dev_Main_Window_picture;
    //unsigned char* fftarray;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float milliseconds_1 = 0, milliseconds_2 = 0, milliseconds_3 = 0, milliseconds_4 = 0;

    hipEventRecord(start);
    hipError_t cudaStatus;
    static int k = 0;
    static char filepath[256];
  
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds_1, start, stop);
    printf("\n/////////\nload img to aplication : %.5f millisec\n", milliseconds_1);

   
    hipEventRecord(start);
    cudaStatus = hipMalloc((void**)&dev_Main_Window_picture, width_A * width_A * sizeof(unsigned char) * 1);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error1;

    }
    cudaStatus = hipMemcpy(dev_Main_Window_picture, dev_a, width_A * width_A * sizeof(unsigned char) * 1, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        // goto Error1;
    }
    cudaStatus = hipMalloc((void**)&fftarray, width_A * width_A * sizeof(hipfftDoubleComplex));
    if (cudaStatus != hipSuccess) {
        //goto Error;
        // MessageBox(NULL, "zarray : hipMalloc failed!", "CudaError", NULL);
      //  goto Error1;
    }
    int thread = 0;
    if (width_A < 1024)
    {
        thread = width_A;
    }
    else
    {
        thread = 1024;
    }

    dim3    grids((width_A + 32 - 1) / 32, (width_A + 32 - 1) / 32);
    dim3    threads(32, 32);

    /*Img_to_Complex << <grids, threads >> > (zarray, dev_Main_Window_picture, texWidth);*/
    Img_to_cufftComplex << <grids, threads >> > (fftarray, dev_Main_Window_picture, width_A);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds_2, start, stop);
    printf("create\load to complex : %.5f millisec\n", milliseconds_2);

    //hipEventRecord(start);
    MYBPF(bufferIMG, width_A);

    //hipEventRecord(stop);
    //hipEventSynchronize(stop);
    //hipEventElapsedTime(&milliseconds_3, start, stop);
    //printf("BPF : %.5f millisec\n", milliseconds_3);

    hipEventRecord(start);
    /*   stbi_image_free(pixels_Gal);*/
    dim3 grids1(DIM / 16, DIM / 16);
    dim3 threads1(16, 16);
    //kernel << <grids1, threads1 >> > (pixels, ticks, texWidth_A, texHeight_A, texWidth_B, texHeight_B, dev_a, dev_b);
  
    
    DrawPicABSAfterBPFRange << <grids1, threads1 >> > (bufferIMG, fftarray, width_A, Min, Max);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds_4, start, stop);
    printf("Pan/crop/scaling : %.5f millisec\n", milliseconds_4);
    printf("full time without draw : %.5f millisec\n", milliseconds_4 + milliseconds_3 + milliseconds_2 + milliseconds_1);
    //Error1:
   // stbi_image_free(dev_a);
    hipFree(fftarray);
    hipFree(dev_Main_Window_picture);

}



